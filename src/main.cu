#include "hip/hip_runtime.h"
#include <GL/glut.h>
#include <iostream>
#include "u.hpp"

__global__ void Hello(){
    printf("Hello From Cuda~ \n");
}
void DrawGrid(int HALF_GRID_SIZE)
{
    glBegin(GL_LINES);
    glColor3f(0.75f, 0.75f, 0.75f);
    for(int i=-HALF_GRID_SIZE;i<=HALF_GRID_SIZE;i++)
    {
        glVertex3f((float)i,0,(float)-HALF_GRID_SIZE);
        glVertex3f((float)i,0,(float)HALF_GRID_SIZE);

        glVertex3f((float)-HALF_GRID_SIZE,0,(float)i);
        glVertex3f((float)HALF_GRID_SIZE,0,(float)i);
    }
    glEnd();

    //call it like this
    DrawGrid(10);
}

void displayMe(void)
{
    glClear(GL_COLOR_BUFFER_BIT);
    glBegin(GL_POLYGON);
        glVertex3f(0.5, 0.0, 0.5);
        glVertex3f(0.5, 0.0, 0.0);
        glVertex3f(0.0, 0.5, 0.0);
        glVertex3f(0.0, 0.0, 0.5);
    glEnd();
    glFlush();
}
 
int main(int argc, char** argv)
{
    
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_SINGLE);
    glutInitWindowSize(800, 600);
    glutInitWindowPosition(400, 400);
    glutCreateWindow("Hello world!");
    glutDisplayFunc(displayMe);
    glutMainLoop();
    return 0;
}